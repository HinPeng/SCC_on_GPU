#include "hip/hip_runtime.h"
#include "bfs.hpp"
#include "bfs_kernels.cuh"
#include "compaction.cuh"
#include <stdio.h>

extern __device__  unsigned terminate_d;
extern __device__ unsigned numActiveThreads;

__host__
void setUInt(unsigned *address, unsigned value) {
    gpuErrchk(hipMemcpy(address, &value, sizeof(unsigned), hipMemcpyHostToDevice));
}

// If you are going to debug
__global__
void output(int N, unsigned *ptr) {
    for (int i = 0; i < N; ++i) {
        printf("%u ", ptr[i]);
    }
    printf("\n");
}

__host__
void BFS(vector<unsigned> &V, vector<unsigned> &E, unsigned sourceVertex, std::vector<unsigned> & distances) {

    assert(sizeof(unsigned) == 4);
    
    distances.clear();
    distances.resize(N);

    // Memory allocation and setup

    unsigned *d_V, *d_E;
    unsigned *d_F, *d_X, *d_C, *d_Fu;
    unsigned *activeMask, *prefixSums;

    size_t memSize = (N + 1) * sizeof(unsigned);
    
    gpuErrchk(hipMalloc(&d_F, memSize));
    gpuErrchk(hipMemset(d_F, FALSE, memSize));
    setUInt(d_F + sourceVertex, TRUE); // add source to frontier

    gpuErrchk(hipMalloc(&d_X, memSize));
    gpuErrchk(hipMemset(d_X, FALSE, memSize));
    setUInt(d_X + sourceVertex, TRUE); // set source as visited

    gpuErrchk(hipMalloc(&d_C, memSize));
    gpuErrchk(hipMemset(d_C, 255, memSize)); // set "infinite" distance
    setUInt(d_C + sourceVertex, FALSE); // set zero distance to source

    gpuErrchk(hipMalloc(&d_Fu, memSize));
    gpuErrchk(hipMemset(d_Fu, FALSE, memSize));

    gpuErrchk(hipMalloc(&d_V, memSize));
    gpuErrchk(hipMemcpy(d_V, V.data(), memSize, hipMemcpyHostToDevice));

    size_t memSizeE = M * sizeof(unsigned);
    gpuErrchk(hipMalloc(&d_E, memSizeE));
    gpuErrchk(hipMemcpy(d_E, E.data(), memSizeE, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&activeMask, memSize));
    setUInt(activeMask + 0, sourceVertex); // set thread #source as active

    unsigned numActiveThreadsHost = 1;
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(numActiveThreads), &numActiveThreadsHost, sizeof(unsigned)));

    gpuErrchk(hipMalloc(&prefixSums, memSize));
    preallocBlockSums(N + 1);

    // Main loop

    const size_t prefixSumGridSize = 
        (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    while (true) {

        // Terminate <- TRUE
        unsigned terminateHost = TRUE;
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(terminate_d), &terminateHost, sizeof(unsigned)));

        // Kernel 1: need to assign ACTIVE vertices to SIMD lanes (threads)
        //gpuErrchk(hipMemcpyFromSymbol(&numActiveThreadsHost, HIP_SYMBOL(numActiveThreads), sizeof(unsigned)));
        const size_t gridSizeK1 = 
            (numActiveThreadsHost + BLOCK_SIZE - 1) / BLOCK_SIZE;

        // launch kernel 1
        BFSKernel1 <<<gridSizeK1, BLOCK_SIZE>>> (N, activeMask, d_V, d_E, d_F, d_X, d_C, d_Fu);
        //gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // Kernel 2: need to assign ALL vertices to SIMD lanes
        const size_t gridSizeK2 =
            (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

        // launch kernel 2
        BFSKernel2 <<<gridSizeK2, BLOCK_SIZE>>> (N, d_F, d_X, d_Fu);
        //gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        gpuErrchk(hipMemcpyFromSymbol(&terminateHost, HIP_SYMBOL(terminate_d), sizeof(unsigned)));

        if (terminateHost) {
            break;
        } else {
            // Get prefix sums of F
            prescanArray(prefixSums, d_F, N + 1);
            //hipMemcpy(&numActiveThreads, prefixSums + N, sizeof(unsigned), hipMemcpyDeviceToDevice);
            hipMemcpy(&numActiveThreadsHost, prefixSums + N, sizeof(unsigned), hipMemcpyDeviceToHost);
            gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(numActiveThreads), &numActiveThreadsHost, sizeof(unsigned)));


            /*gpuErrchk(hipMemcpyFromSymbol(&numActiveThreadsHost, HIP_SYMBOL(numActiveThreads), sizeof(unsigned)));
            printf("%u\n", numActiveThreadsHost);*/
            
            const size_t gridSizeCompaction = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
            compactSIMD <<<gridSizeCompaction, BLOCK_SIZE>>> (N, prefixSums, activeMask, BLOCK_SIZE);
            //gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());

            //gpuErrchk(hipPeekAtLastError());
            //gpuErrchk(hipDeviceSynchronize());
        }
    }

    // Download result

    gpuErrchk(hipMemcpy(distances.data(), d_X, memSize-sizeof(unsigned), hipMemcpyDeviceToHost));

    // Free memory

    gpuErrchk(hipFree(d_F));
    gpuErrchk(hipFree(d_X));
    gpuErrchk(hipFree(d_C));
    gpuErrchk(hipFree(d_Fu));
    gpuErrchk(hipFree(d_V));
    gpuErrchk(hipFree(d_E));
    gpuErrchk(hipFree(activeMask));
    deallocBlockSums();
    gpuErrchk(hipFree(prefixSums));
}


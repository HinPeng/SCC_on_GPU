#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <map>
#include <iterator>
#include <algorithm>
#include <assert.h>
#include <time.h>

using namespace std;


const int N = 2394385;
const int M = 5021410;
const int BLOCK_SIZE = 256;

const unsigned MAX_NUM = 4294967295;

const int NUM_BANKS = 16;
const int LOG_NUM_BANKS = 4;
const string fn("WikiTalk.txt");
//const string fn("soc-LiveJournal1.txt");

unsigned **scanBlockSums;
unsigned numEltsAllocated = 0;
unsigned numLevelsAllocated = 0;

__device__  unsigned Mterminate;
//__managed__ unsigned numActiveThreads;
__device__ unsigned numActiveThreads;
__device__ unsigned *range;
__device__ unsigned pivot;

#define FALSE 0u
#define  TRUE 1u

#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS)
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "Error: %s\nFile %s, line %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

inline
bool isPowerOfTwo(int n) {
	return (n & (n - 1)) == 0;
}

inline
int floorPow2(int n) {
	int exp;
	frexp((float)n, &exp);
	return 1 << (exp - 1);
}

template <bool isNP2>
__device__
void loadSharedChunkFromMem(unsigned *s_data, const unsigned *idata, int n, int baseIndex, int& ai, int& bi, int& mem_ai, int& mem_bi, int& bankOffsetA, int& bankOffsetB);
template <bool isNP2>
__device__
void storeSharedChunkToMem(unsigned* odata, const unsigned* s_data, int n, int ai, int bi, int mem_ai, int mem_bi, int bankOffsetA, int bankOffsetB);
template <bool storeSum>
__device__
void clearLastElement(unsigned* s_data, unsigned *blockSums, int blockIndex);
__device__
unsigned int buildSum(unsigned *s_data);
__device__
void scanRootToLeaves(unsigned *s_data, unsigned int stride);
template <bool storeSum>
__device__
void prescanBlock(unsigned *data, int blockIndex, unsigned *blockSums);
template <bool storeSum, bool isNP2>
__global__
void prescan(unsigned *odata, const unsigned *idata, unsigned *blockSums, int n, int blockIndex, int baseIndex);
__global__
void uniformAdd(unsigned *data, unsigned *uniforms, int n, int blockOffset, int baseIndex);

__global__
void BFSKernel1(
size_t graphSize, unsigned *activeMask, unsigned *V, unsigned *E,
unsigned *F, unsigned *X,unsigned *Fu);
__global__
void BFSKernel2(size_t graphSize, unsigned *F, unsigned *X, unsigned *Fu);
__global__
void TRIMKernel(size_t graphSize,unsigned subnum, unsigned *V,unsigned *E);
__global__
void getActiveMaskTemp(size_t graphSize, unsigned *F, unsigned *activeMask);
__global__
void compactSIMD(size_t N, unsigned *prefixSums, unsigned *activeMask, size_t blockSize);
__global__
void PIVOTS_SEL_Kernel(size_t graphSize,unsigned subnum);
__global__
void UpdateKernel(unsigned cur_max_sub,unsigned subnum,unsigned *visF,unsigned *visB,unsigned *scc);

__host__
void setUInt(unsigned *address, unsigned value);
__host__
void Graphpreproc(const string filename, vector<unsigned> &VF, vector<unsigned> &EF, vector<unsigned> &VB, vector<unsigned> &EB);
__host__
void BFS(vector<unsigned> &V, vector<unsigned> &E, unsigned sourceVertex,vector<unsigned> &visited);
__host__
void TRIMMING(unsigned index_s,unsigned index_e, vector<unsigned> &V, vector<unsigned> &E);
__host__
void PIVOTS_SEL(unsigned subnum,unsigned &pivot_h,unsigned &termin);
__host__
void Update(unsigned cur_max_sub,unsigned subnum,vector<unsigned> &visF,vector<unsigned> &visB,vector<unsigned> &scc,unsigned &termin);
__host__
void preallocBlockSums(unsigned maxNumElements);
__host__
void deallocBlockSums();
__host__
void prescanArrayRecursive(unsigned *outArray, const unsigned *inArray, int numElements, int level);
__host__
void prescanArray(unsigned *outArray, unsigned *inArray, int numElements);

int main()
{
	vector<unsigned> VF, EF, VB, EB, visF, visB, scc;
	vector<unsigned>::iterator itr;

	ofstream out("out1.txt");
	if(!out){  
    	cout << "Unable to open outfile";  
        exit(1); // terminate with error  
    }
  

	bool flag = true;

	unsigned pivot_h;
	unsigned termin;

	unsigned *d_r;

	VF.reserve(N + 1);
	EF.reserve(M);
	VB.reserve(N + 1);
	EB.reserve(M);
	Graphpreproc(fn, VF, EF, VB, EB);

	unsigned st = 0;
	unsigned ed = 0;
	unsigned cur_max_sub = 0;

	long trim = 0;
	long bfs = 0;
	long pivsel = 0;
	long upd = 0;

	clock_t start=clock();

	gpuErrchk(hipMalloc(&d_r, N*sizeof(unsigned)));
	gpuErrchk(hipMemset(d_r, FALSE, N*sizeof(unsigned)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(range), &d_r, sizeof(unsigned *),size_t(0), hipMemcpyHostToDevice));

	while(flag){
		flag = false;
		clock_t start_trim=clock();
		TRIMMING(st,ed,VF,EF);
		TRIMMING(st,ed,VB,EB);
		clock_t finish_trim=clock();
		trim += (finish_trim - start_trim);
		for(unsigned i = st; i <= ed; i++){
			clock_t start_pivsel=clock();
			PIVOTS_SEL(i,pivot_h,termin);
			clock_t finish_pivsel=clock();
			pivsel += (finish_pivsel - start_pivsel);
			if(termin)
				continue;

			clock_t start_bfs=clock();
			BFS(VF, EF, pivot_h, visF);
			BFS(VB, EB, pivot_h, visB);
			clock_t finish_bfs=clock();
			bfs += (finish_bfs - start_bfs);

			clock_t start_upd=clock();
			Update(cur_max_sub,i,visF,visB,scc,termin);
			clock_t finish_upd=clock();
			upd += (finish_upd - start_upd);

			unsigned j = 0;
			for(itr = scc.begin();itr!=scc.end();itr++){
				if(*itr == 1){
					j++;
				}
			}
			out<<j<<endl;
			cur_max_sub += 3;
			if(termin == FALSE)
				flag = true;
		}
		st = ed + 1;
		ed = cur_max_sub;
	}
	clock_t finish=clock();
	printf("time elapsed:%.2fs\n",(double)(finish-start)/1.0e6);
	printf("TRIM time elapsed:%.2fs\n",(double)(trim)/1.0e6);
	printf("PIVSEL time elapsed:%.2fs\n",(double)(pivsel)/1.0e6);
	printf("BFS time elapsed:%.2fs\n",(double)(bfs)/1.0e6);
	printf("UPD time elapsed:%.2fs\n",(double)(upd)/1.0e6);
	printf("%u\n",ed -3);

	out.close();
	return 0;
}

__host__
void setUInt(unsigned *address, unsigned value) {
	gpuErrchk(hipMemcpy(address, &value, sizeof(unsigned), hipMemcpyHostToDevice));
}

__host__
void Graphpreproc(const string filename, vector<unsigned> &VF, vector<unsigned> &EF, vector<unsigned> &VB, vector<unsigned> &EB)
{
	ifstream in_f;
	vector<unsigned> t;
	vector<unsigned>::iterator itr;
	multimap<const unsigned, unsigned> m;
	multimap<const unsigned, unsigned>::iterator mitr;
	unsigned count = 0;
	in_f.open(filename.c_str(), ios::in);
	while (!in_f.eof()){
		string temp, s1, s2;
		stringstream ss1, ss2;
		unsigned t1, t2;
		getline(in_f, temp);
		if(temp.length() == 0)
			continue;
		if (*(temp.begin()) == '#')
			continue;
		s1 = string(temp, 0, temp.find_first_of('\t'));
		s2 = string(temp, temp.find_first_not_of('\t', temp.find_first_of('\t')), temp.find_last_not_of('\t'));
		ss1 << s1;
		ss1 >> t1;
		ss2 << s2;
		ss2 >> t2;
		t.push_back(t1);
		m.insert(make_pair(t2, t1));
		EF.push_back(t2);
	}

	itr = t.begin();
	VF.push_back(0);
	for (int i = 0; i < N - 1; i++){
		while ((itr != t.end()) && (*itr == i)){
			count++;
			itr++;
		}
		count += VF.at(i);
		VF.push_back(count);
		count = 0;
	}
	VF.push_back(M);

	mitr = m.begin();
	VB.push_back(0);
	for (int i = 0; i < N - 1; i++){
		while ((mitr != m.end()) && ((mitr->first) == i)){
			count++;
			mitr++;
		}
		count += VB.at(i);
		VB.push_back(count);
		count = 0;
	}
	VB.push_back(M);
	for (mitr = m.begin(); mitr != m.end(); mitr++){
		EB.push_back(mitr->second);
	}
}

__global__
void BFSKernel1(
size_t graphSize, unsigned *activeMask, unsigned *V, unsigned *E,
unsigned *F, unsigned *X,unsigned *Fu) {

	unsigned activeMaskIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	// If vertex is active at current iteration
	if (activeMaskIdx < numActiveThreads) {

		unsigned v = activeMask[activeMaskIdx];

		// Remove v from current frontier
		F[v] = FALSE;

		// Iterate over v's neighbors
		for (unsigned edge = V[v]; edge < V[v + 1]; ++edge) {
			unsigned neighbor = E[edge];

			// If neighbor wasn't visited
			if(range[v] == range[neighbor]){
				if (X[neighbor] == FALSE){
					//C[neighbor] = C[v] + 1;
					Fu[neighbor] = TRUE;
				}
			}
		}
	}
}

__global__
void BFSKernel2(size_t graphSize, unsigned *F, unsigned *X, unsigned *Fu) {

	int v = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	// If vertex v exists and has recently joined the frontier
	if (v < graphSize && Fu[v]) {
		// Copy the new frontier into F
		F[v] = TRUE;
		// Set v as visited
		X[v] = TRUE;
		// Clean up the new frontier
		Fu[v] = FALSE;

		Mterminate = FALSE;
	}
}

__global__
void TRIMKernel(size_t graphSize,unsigned subnum, unsigned *V,unsigned *E)
{
	int v = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	bool elim;

	if((v < graphSize)&&(range[v] == subnum)){
		elim = true;
		for (unsigned edge = V[v]; edge < V[v + 1]; ++edge) {
			unsigned neighbor = E[edge];
			if(range[neighbor] == subnum)
				elim = false;
		}
		if(elim == true){
			range[v] = MAX_NUM;
			Mterminate = FALSE;
		}
	}
}

__global__
void PIVOTS_SEL_Kernel(size_t graphSize,unsigned subnum)
{
	int v = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	if((v < graphSize)&&(range[v] == subnum)){
		pivot = v;
		Mterminate = FALSE;
	}
}

__global__
void UpdateKernel(unsigned cur_max_sub,unsigned subnum,unsigned *visF,unsigned *visB,unsigned *scc)
{
	int v = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	if(range[v] == subnum){
		if(visF[v] == TRUE){
			if(visB[v] == TRUE){
				scc[v] = TRUE;
				range[v] = MAX_NUM;
			}
			else if(visB[v] == FALSE){
				range[v] = cur_max_sub + 1;
				Mterminate = FALSE;
			}
		}
		else if(visF[v] == FALSE){
			if(visB[v] == TRUE){
				range[v] = cur_max_sub + 2;
				Mterminate = FALSE;
			}
			else if(visB[v] == FALSE){
				range[v] = cur_max_sub + 3;
				Mterminate = FALSE;
			}
		}
	}
}

// Very slow but correct "active mask" calculation; for debugging
__global__
void getActiveMaskTemp(size_t graphSize, unsigned *F, unsigned *activeMask) {

	numActiveThreads = 0;
	for (int i = 0; i < graphSize; ++i) {
		if (F[i]) {
			activeMask[numActiveThreads] = i;
			++numActiveThreads;
		}
	}
}

__global__
void compactSIMD(size_t N, unsigned *prefixSums, unsigned *activeMask, size_t blockSize) {

	size_t v = blockIdx.x * blockSize + threadIdx.x;

	if (v < N) {
		// Can possibly be accelerated by using shared memory
		if (prefixSums[v + 1] != prefixSums[v]) {
			activeMask[prefixSums[v]] = v;
		}
	}
}

__host__
void BFS(vector<unsigned> &V, vector<unsigned> &E, unsigned sourceVertex,vector<unsigned> &visited)
{
	assert(sizeof(unsigned) == 4);
	visited.clear();
	visited.resize(N);

	unsigned *d_V, *d_E;
	unsigned *d_F, *d_X, *d_Fu;
	unsigned *activeMask, *prefixSums;
	//unsigned **prefixSums;
	size_t memSize = (N + 1) * sizeof(unsigned);
	size_t memSizeE = M * sizeof(unsigned);

	gpuErrchk(hipMalloc(&d_F, memSize));
	gpuErrchk(hipMemset(d_F, FALSE, memSize));
	setUInt(d_F + sourceVertex, TRUE); // add source to frontier

	gpuErrchk(hipMalloc(&d_X, memSize));
	gpuErrchk(hipMemset(d_X, FALSE, memSize));
	setUInt(d_X + sourceVertex, TRUE); // set source as visited

	//gpuErrchk(hipMalloc(&d_C, memSize));
	//gpuErrchk(hipMemset(d_C, 255, memSize)); // set "infinite" distance
	//setUInt(d_C + sourceVertex, FALSE); // set zero distance to source

	gpuErrchk(hipMalloc(&d_Fu, memSize));
	gpuErrchk(hipMemset(d_Fu, FALSE, memSize));

	gpuErrchk(hipMalloc(&d_V, memSize));
	gpuErrchk(hipMemcpy(d_V, V.data(), memSize, hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc(&d_E, memSizeE));
	gpuErrchk(hipMemcpy(d_E, E.data(), memSizeE, hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc(&activeMask, memSize));
	setUInt(activeMask + 0, sourceVertex); // set thread #source as active

	unsigned numActiveThreadsHost = 1;
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(numActiveThreads), &numActiveThreadsHost, sizeof(unsigned)));

	gpuErrchk(hipMalloc(&prefixSums, memSize));
	preallocBlockSums(N + 1);

	// Main loop

	/*const size_t prefixSumGridSize =
		(N + BLOCK_SIZE - 1) / BLOCK_SIZE;*/

	while (true) {

		// Terminate <- TRUE
		unsigned terminateHost = TRUE;

		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(Mterminate), &terminateHost, sizeof(unsigned)));

		// Kernel 1: need to assign ACTIVE vertices to SIMD lanes (threads)
		gpuErrchk(hipMemcpyFromSymbol(&numActiveThreadsHost, HIP_SYMBOL(numActiveThreads), sizeof(unsigned)));
		const size_t gridSizeK1 =
			(numActiveThreadsHost + BLOCK_SIZE - 1) / BLOCK_SIZE;

		// launch kernel 1
		BFSKernel1 <<<gridSizeK1, BLOCK_SIZE >>> (N, activeMask, d_V, d_E, d_F, d_X,d_Fu);
		//gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		// Kernel 2: need to assign ALL vertices to SIMD lanes
		const size_t gridSizeK2 =
			(N + BLOCK_SIZE - 1) / BLOCK_SIZE;

		// launch kernel 2
		BFSKernel2 << <gridSizeK2, BLOCK_SIZE >> > (N, d_F, d_X, d_Fu);
		//gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());


		gpuErrchk(hipMemcpyFromSymbol(&terminateHost, HIP_SYMBOL(Mterminate), sizeof(unsigned)));

		if (terminateHost) {
			break;
		}
		else {
			// Get prefix sums of F
			prescanArray(prefixSums, d_F, N + 1);
			//hipMemcpy(&numActiveThreads, prefixSums + N, sizeof(unsigned), hipMemcpyDeviceToDevice);
			hipMemcpy(&numActiveThreadsHost, prefixSums + N, sizeof(unsigned), hipMemcpyDeviceToHost);
            gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(numActiveThreads), &numActiveThreadsHost, sizeof(unsigned)));

			const size_t gridSizeCompaction = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
			compactSIMD <<<gridSizeCompaction, BLOCK_SIZE >>> (N, prefixSums, activeMask, BLOCK_SIZE);
			//gpuErrchk(hipPeekAtLastError());
			//getActiveMaskTemp<<<1,1>>>(N,d_F,activeMask);
			gpuErrchk(hipDeviceSynchronize());

			//gpuErrchk(hipPeekAtLastError());
		}
	}

	// Download result

	//gpuErrchk(hipMemcpy(distances.data(), d_C, memSize - sizeof(unsigned), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(visited.data(), d_X, memSize - sizeof(unsigned), hipMemcpyDeviceToHost));

	// Free memory

	gpuErrchk(hipFree(d_F));
	gpuErrchk(hipFree(d_X));
	gpuErrchk(hipFree(d_Fu));
	gpuErrchk(hipFree(d_V));
	gpuErrchk(hipFree(d_E));
	gpuErrchk(hipFree(activeMask));
	deallocBlockSums();
	gpuErrchk(hipFree(prefixSums));
}

__host__
void TRIMMING(unsigned index_s,unsigned index_e,vector<unsigned> &V, vector<unsigned> &E)
{
	unsigned *d_V, *d_E;

	size_t memSize = (N + 1) * sizeof(unsigned);
	size_t memSizeE = M * sizeof(unsigned);

	gpuErrchk(hipMalloc(&d_V, memSize));
	gpuErrchk(hipMemcpy(d_V, V.data(), memSize, hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc(&d_E, memSizeE));
	gpuErrchk(hipMemcpy(d_E, E.data(), memSizeE, hipMemcpyHostToDevice));

	const size_t gridSizeK2 =
			(N + BLOCK_SIZE - 1) / BLOCK_SIZE;

	hipStream_t stream[index_e-index_s+1];
	for (int i = 0; i < index_e-index_s+1; ++i)
    	hipStreamCreate(&stream[i]);

    for (int i = 0; i < index_e-index_s+1; ++i){
		while (true) {

		// Terminate <- TRUE
			unsigned terminateHost = TRUE;

			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(Mterminate), &terminateHost, sizeof(unsigned)));

		// Kernel 2: need to assign ALL vertices to SIMD lanes
		
		// launch kernel 2
			TRIMKernel <<<gridSizeK2, BLOCK_SIZE, 0, stream[i] >>> (N, index_s+i,d_V, d_E);
		//gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());


			gpuErrchk(hipMemcpyFromSymbol(&terminateHost, HIP_SYMBOL(Mterminate), sizeof(unsigned)));

			if (terminateHost) {
			break;
			}
		}
	}

	for (int i = 0; i < index_e-index_s+1; ++i)
    	hipStreamDestroy(stream[i]);

	gpuErrchk(hipFree(d_V));
	gpuErrchk(hipFree(d_E));
}

__host__
void PIVOTS_SEL(unsigned subnum,unsigned &pivot_h,unsigned &termin)
{
	const size_t gridSizeK2 =
			(N + BLOCK_SIZE - 1) / BLOCK_SIZE;

	termin = TRUE;

	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(Mterminate), &termin, sizeof(unsigned)));

	PIVOTS_SEL_Kernel<<<gridSizeK2, BLOCK_SIZE>>>(N,subnum);

	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpyFromSymbol(&pivot_h, HIP_SYMBOL(pivot), sizeof(unsigned)));
	gpuErrchk(hipMemcpyFromSymbol(&termin, HIP_SYMBOL(Mterminate), sizeof(unsigned)));
}

__host__
void Update(unsigned cur_max_sub,unsigned subnum,vector<unsigned> &visF,vector<unsigned> &visB,vector<unsigned> &scc,unsigned &termin)
{
	scc.clear();
	scc.resize(N);

	unsigned *d_vf,*d_vb;
	unsigned *d_scc;

	size_t memSize = N * sizeof(unsigned);

	gpuErrchk(hipMalloc(&d_vf, memSize));
	gpuErrchk(hipMemcpy(d_vf, visF.data(), memSize, hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc(&d_vb, memSize));
	gpuErrchk(hipMemcpy(d_vb, visB.data(), memSize, hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc(&d_scc, memSize));
	gpuErrchk(hipMemset(d_scc, FALSE, memSize));

	// Terminate <- TRUE
	termin = TRUE;

	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(Mterminate), &termin, sizeof(unsigned)));

	const size_t gridSizeK2 =
			(N + BLOCK_SIZE - 1) / BLOCK_SIZE;

	UpdateKernel<<<gridSizeK2, BLOCK_SIZE>>>(cur_max_sub,subnum,d_vf,d_vb,d_scc);

	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpy(scc.data(), d_scc, memSize, hipMemcpyDeviceToHost));

	gpuErrchk(hipMemcpyFromSymbol(&termin, HIP_SYMBOL(Mterminate), sizeof(unsigned)));

	gpuErrchk(hipFree(d_vf));
	gpuErrchk(hipFree(d_vb));
	gpuErrchk(hipFree(d_scc));
}

__host__
void preallocBlockSums(unsigned maxNumElements) {
	numEltsAllocated = maxNumElements;

	unsigned blockSize = BLOCK_SIZE;
	unsigned numElts = maxNumElements;

	int level = 0;

	do {
		unsigned numBlocks =
			max(1, (int)ceil((float)numElts / (2.f * blockSize)));
		if (numBlocks > 1) {
			level++;
		}
		numElts = numBlocks;
	} while (numElts > 1);

	scanBlockSums = (unsigned**)malloc(level * sizeof(unsigned*));
	numLevelsAllocated = level;

	numElts = maxNumElements;
	level = 0;

	do {
		unsigned numBlocks =
			max(1, (int)ceil((float)numElts / (2.f * blockSize)));
		if (numBlocks > 1) {
			gpuErrchk(hipMalloc(&scanBlockSums[level++], numBlocks * sizeof(unsigned)));
		}
		numElts = numBlocks;
	} while (numElts > 1);
}

__host__
void deallocBlockSums() {
	for (unsigned i = 0; i < numLevelsAllocated; i++) {
		hipFree(scanBlockSums[i]);
	}

	free(scanBlockSums);

	scanBlockSums = 0;
	numEltsAllocated = 0;
	numLevelsAllocated = 0;
}

__host__
void prescanArrayRecursive(unsigned *outArray,
const unsigned *inArray,
int numElements,
int level) {

	unsigned blockSize = BLOCK_SIZE;
	unsigned numBlocks =
		max(1, (int)ceil((float)numElements / (2.f * blockSize)));
	unsigned numThreads;

	if (numBlocks > 1)
		numThreads = blockSize;
	else if (isPowerOfTwo(numElements))
		numThreads = numElements / 2;
	else
		numThreads = floorPow2(numElements);

	unsigned numEltsPerBlock = numThreads * 2;

	unsigned numEltsLastBlock =
		numElements - (numBlocks - 1) * numEltsPerBlock;
	unsigned numThreadsLastBlock = max(1u, numEltsLastBlock / 2);
	unsigned np2LastBlock = 0;
	unsigned sharedMemLastBlock = 0;

	if (numEltsLastBlock != numEltsPerBlock) {
		np2LastBlock = 1;

		if (!isPowerOfTwo(numEltsLastBlock))
			numThreadsLastBlock = floorPow2(numEltsLastBlock);

		unsigned extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
		sharedMemLastBlock =
			sizeof(unsigned)* (2 * numThreadsLastBlock + extraSpace);
	}

	// Avoid shared memory bank conflicts
	unsigned extraSpace = numEltsPerBlock / NUM_BANKS;
	unsigned sharedMemSize =
		sizeof(unsigned)* (numEltsPerBlock + extraSpace);

	dim3 grid(max(1u, numBlocks - np2LastBlock), 1, 1);
	dim3 threads(numThreads, 1, 1);

	// Main action

	if (numBlocks > 1) {
		prescan<true, false> << < grid, threads, sharedMemSize >> > (
			outArray, inArray, scanBlockSums[level], numThreads * 2, 0, 0);

		if (np2LastBlock) {
			prescan<true, true> << < 1, numThreadsLastBlock, sharedMemLastBlock >> > (
				outArray, inArray, scanBlockSums[level], numEltsLastBlock,
				numBlocks - 1, numElements - numEltsLastBlock);
		}

		prescanArrayRecursive(scanBlockSums[level], scanBlockSums[level], numBlocks, level + 1);

		uniformAdd << < grid, threads >> > (
			outArray, scanBlockSums[level], numElements - numEltsLastBlock, 0, 0);

		if (np2LastBlock) {
			uniformAdd << <1, numThreadsLastBlock >> > (
				outArray, scanBlockSums[level], numEltsLastBlock,
				numBlocks - 1, numElements - numEltsLastBlock);
		}
	}
	else if (isPowerOfTwo(numElements)) {
		prescan<false, false> << <grid, threads, sharedMemSize >> > (
			outArray, inArray, 0, numThreads * 2, 0, 0);
	}
	else {
		prescan<false, true> << <grid, threads, sharedMemSize >> > (
			outArray, inArray, 0, numElements, 0, 0);
	}
}

__host__
void prescanArray(unsigned *outArray, unsigned *inArray, int numElements) {
	prescanArrayRecursive(outArray, inArray, numElements, 0);
}

template <bool isNP2>
__device__ void loadSharedChunkFromMem(unsigned *s_data,
	const unsigned *idata,
	int n, int baseIndex,
	int& ai, int& bi,
	int& mem_ai, int& mem_bi,
	int& bankOffsetA, int& bankOffsetB) {
	int thid = threadIdx.x;
	mem_ai = baseIndex + threadIdx.x;
	mem_bi = mem_ai + blockDim.x;

	ai = thid;
	bi = thid + blockDim.x;

	bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	bankOffsetB = CONFLICT_FREE_OFFSET(bi);

	s_data[ai + bankOffsetA] = idata[mem_ai];

	if (isNP2) {
		s_data[bi + bankOffsetB] = (bi < n) ? idata[mem_bi] : 0;
	}
	else {
		s_data[bi + bankOffsetB] = idata[mem_bi];
	}
}

template <bool isNP2>
__device__
void storeSharedChunkToMem(unsigned* odata,
const unsigned* s_data,
int n,
int ai, int bi,
int mem_ai, int mem_bi,
int bankOffsetA, int bankOffsetB) {
	__syncthreads();

	odata[mem_ai] = s_data[ai + bankOffsetA];
	if (isNP2) {
		if (bi < n)
			odata[mem_bi] = s_data[bi + bankOffsetB];
	}
	else {
		odata[mem_bi] = s_data[bi + bankOffsetB];
	}
}

template <bool storeSum>
__device__
void clearLastElement(unsigned* s_data,
unsigned *blockSums,
int blockIndex) {
	if (threadIdx.x == 0) {
		int index = (blockDim.x << 1) - 1;
		index += CONFLICT_FREE_OFFSET(index);

		if (storeSum) {
			blockSums[blockIndex] = s_data[index];
		}

		s_data[index] = 0;
	}
}

__device__
unsigned int buildSum(unsigned *s_data) {
	unsigned int thid = threadIdx.x;
	unsigned int stride = 1;

	for (int d = blockDim.x; d > 0; d >>= 1) {
		__syncthreads();

		if (thid < d) {
			int i = __mul24(__mul24(2, stride), thid);
			int ai = i + stride - 1;
			int bi = ai + stride;

			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			s_data[bi] += s_data[ai];
		}

		stride *= 2;
	}

	return stride;
}

__device__
void scanRootToLeaves(unsigned *s_data, unsigned int stride) {
	unsigned int thid = threadIdx.x;

	for (int d = 1; d <= blockDim.x; d *= 2) {
		stride >>= 1;

		__syncthreads();

		if (thid < d)
		{
			int i = __mul24(__mul24(2, stride), thid);
			int ai = i + stride - 1;
			int bi = ai + stride;

			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			unsigned t = s_data[ai];
			s_data[ai] = s_data[bi];
			s_data[bi] += t;
		}
	}
}

template <bool storeSum>
__device__
void prescanBlock(unsigned *data, int blockIndex, unsigned *blockSums) {
	int stride = buildSum(data);
	clearLastElement<storeSum>(data, blockSums,
		(blockIndex == 0) ? blockIdx.x : blockIndex);
	scanRootToLeaves(data, stride);
}

template <bool storeSum, bool isNP2>
__global__
void prescan(unsigned *odata,
const unsigned *idata,
unsigned *blockSums,
int n,
int blockIndex,
int baseIndex) {
	int ai, bi, mem_ai, mem_bi, bankOffsetA, bankOffsetB;
	extern __shared__ unsigned s_data[];

	loadSharedChunkFromMem<isNP2>(s_data, idata, n,
		(baseIndex == 0) ?
		__mul24(blockIdx.x, (blockDim.x << 1)) : baseIndex,
		ai, bi, mem_ai, mem_bi,
		bankOffsetA, bankOffsetB);

	prescanBlock<storeSum>(s_data, blockIndex, blockSums);

	storeSharedChunkToMem<isNP2>(odata, s_data, n,
		ai, bi, mem_ai, mem_bi,
		bankOffsetA, bankOffsetB);
}

__global__
void uniformAdd(unsigned *data,
unsigned *uniforms,
int n,
int blockOffset,
int baseIndex) {
	__shared__ unsigned uni;
	if (threadIdx.x == 0)
		uni = uniforms[blockIdx.x + blockOffset];

	unsigned int address = __mul24(blockIdx.x, (blockDim.x << 1)) + baseIndex + threadIdx.x;

	__syncthreads();

	data[address] += uni;
	data[address + blockDim.x] += (threadIdx.x + blockDim.x < n) * uni;
}